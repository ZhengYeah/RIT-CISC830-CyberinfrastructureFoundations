// Enumberate x to compute A^x % M using Binary Exponentiation
// Split [M] into [0, M/2] and [M/2, M] then use two CUDA


#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <omp.h>

__device__ void fast_power(long long base, long long exp, long long B,
                           long long M, long long *result) {
  long long exp_backup = exp;
  __int128_t power_res = 1, base_tmp = base;
  while (exp > 0) {
    if (exp & 1)
      power_res = power_res * base_tmp % M;
    base_tmp = base_tmp * base_tmp % M;
    exp >>= 1;
  }
  if (power_res == B) {
    result[0] = exp_backup;
  }
}

__global__ void discretelog_cuda(long long base, long long B, long long M_start, long long M_end, long long M, long long *result) {
  long long id = M_start + blockIdx.x * blockDim.x + threadIdx.x;
  if (id < M_end) {
    fast_power(base, id, B, M, result);
  }  
}

__global__ void initialize_flag(int *stop_flag) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *stop_flag = 0;
    }
}

int main(int argc, char **argv) {
  int num_GPU = 8;
  long long A = 75052, B = 2375654730, M = 10000000019;


  long long *result_cpu = (long long*) malloc(num_GPU * sizeof(long long)); 
  long long *result_cuda[num_GPU];

  for (int i = 0; i < num_GPU; ++i) {
    hipSetDevice(i);
    hipMalloc((void **)&result_cuda[i], sizeof(long long));
  }

  int BLOCK_SIZE = 1024;
  long long gridSize = (long long) M / BLOCK_SIZE + 1;

  for (int i = 0; i < num_GPU; ++i) {
    hipSetDevice(i);
    discretelog_cuda<<<gridSize, BLOCK_SIZE>>>(A, B, M/num_GPU*i, M/num_GPU*(i+1), M, result_cuda[i]);
    hipMemcpy(result_cpu + i, result_cuda[i], sizeof(long long), hipMemcpyDeviceToHost);
    if (result_cpu[i] != 0) {
      printf("%lld\n", result_cpu[i]);
    }
  }

  for (int i = 0; i < num_GPU; ++i) {
    hipSetDevice(i);
    hipFree(result_cuda[i]);
  }
  free(result_cpu);
  return 0;
}
